#include "hip/hip_runtime.h"
#include "relu.cuh"

// -------------------------------------- FP32 -------------------------------------- 
// Relu x: N, y: N y=max(0, x)
// grid(N/256), block(K=256)
__global__ void relu_f32_kernel(float* x, float* y, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        y[idx] = fmaxf(0.0f, x[idx]);
    }
}

// Relu x: N, y: N y=max(0,x) Vec4
// grid(N/256/4), block(256/4)
__global__ void relu_f32x4_kernel(float* x, float* y, int N) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    if (idx < N) {
        float4 reg_x = FLOAT4(x[idx]);
        float4 reg_y;
        reg_y.x = fmaxf(0.0f, reg_x.x);
        reg_y.y = fmaxf(0.0f, reg_x.y);
        reg_y.z = fmaxf(0.0f, reg_x.z);
        reg_y.w = fmaxf(0.0f, reg_x.w);
        FLOAT4(y[idx]) = reg_y;
    }
}

// -------------------------------------- FP16 -------------------------------------- 
__global__ void relu_f16_kernel(half* x, half* y, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        y[idx] = __hmax(__float2half(0.0f), x[idx]);
    }
}

__global__ void relu_f16x2_kernel(half* x, half* y, int N) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    if (idx < N) {
        half2 reg_x = HALF2(x[idx]);
        half2 reg_y = HALF2(y[idx]);
        reg_y.x = __hmax(__float2half(0.0f), reg_x.x);
        reg_y.y = __hmax(__float2half(0.0f), reg_x.y);
        HALF2(y[idx]) = reg_y;
    }
}

__global__ void relu_f16x8_kernel(half* x, half* y, int N) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
    // manual unroll and improve L2 cache hit rate.
    // Only   L2 cache: load 32  bytes in 1 memory issue (default)
    // Enable L1 cache: load 128 bytes in 1 memory issue (-Xptxas -dlcm=ca)
    // why try fp16x8 within 1 threads? ref: https://zhuanlan.zhihu.com/p/641639133
    // 0. first, tid_0 load 32 bytes in 1 memory issue and cache data into L2 cache.
    // 1. then, tid_1,...,tid_3 hit L2 cache and load data from L2 cache directly.
    half2 reg_x_0 = HALF2(x[idx + 0]);
    half2 reg_x_1 = HALF2(x[idx + 2]);
    half2 reg_x_2 = HALF2(x[idx + 4]);
    half2 reg_x_3 = HALF2(x[idx + 6]);
    half2 reg_y_0, reg_y_1, reg_y_2, reg_y_3;
    reg_y_0.x = __hmax(__float2half(0.0f), reg_x_0.x);
    reg_y_0.y = __hmax(__float2half(0.0f), reg_x_0.y);
    reg_y_1.x = __hmax(__float2half(0.0f), reg_x_1.x);
    reg_y_1.y = __hmax(__float2half(0.0f), reg_x_1.y);
    reg_y_2.x = __hmax(__float2half(0.0f), reg_x_2.x);
    reg_y_2.y = __hmax(__float2half(0.0f), reg_x_2.y);
    reg_y_3.x = __hmax(__float2half(0.0f), reg_x_3.x);
    reg_y_3.y = __hmax(__float2half(0.0f), reg_x_3.y);
    if ((idx + 0) < N) { HALF2(y[idx + 0]) = reg_y_0; }
    if ((idx + 2) < N) { HALF2(y[idx + 2]) = reg_y_1; }
    if ((idx + 4) < N) { HALF2(y[idx + 4]) = reg_y_2; }
    if ((idx + 6) < N) { HALF2(y[idx + 6]) = reg_y_3; }
}

__global__ void relu_f16x8_pack_kernel(half* x, half* y, int N) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
    const half2 z2 = {__float2half(0.0f), __float2half(0.0f)};
    // temporary register(memory), .local space in ptx, addressable
    half pack_x[8], pack_y[8];  // 8x16 bits=128 bits.
    // reinterpret_cast as float4 and load 128 bits in 1 memory issue.
    LDST128BITS(pack_x[0]) = LDST128BITS(x[idx]);     // load 128 bits

    #pragma unroll
    for (int i = 0; i < 8; i += 2) {
        // __hmax2 for half2 x 4
        HALF2(pack_y[i]) = __hmax2(HALF2(pack_x[i]), z2);
    }

    // reinterpret as float4 and store 128 bits in 1 memory issue.
    if ((idx + 7) < N) {
        LDST128BITS(y[idx]) = LDST128BITS(pack_y[0]);
    }
}

