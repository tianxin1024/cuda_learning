// link: https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
// link: https://stackoverflow.com/questions/75883944/wrapping-the-cuda-kernel-function-with-template
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void reduce1(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];

    // each thread loads one element from global to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = g_idata[i];
    __syncthreads();

    // do reduction in shared memory
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}

__global__ void reduce2(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];

    // each thread loads one element from global to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = g_idata[i];
    __syncthreads();

    // do reduction in shared memory
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;

        if (index < blockDim.x) {
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}

__global__ void reduce3(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];

    // each thread loads one element from global to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = g_idata[i];
    __syncthreads();

    // do reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}

__global__ void reduce4(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];

    // perform first level of reduction
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
    __syncthreads();

    // do reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}

__device__ void warpReduce5(volatile int *sdata, int tid) {
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}

__global__ void reduce5(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];

    // perform first level of reduction
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
    __syncthreads();

    // do reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid < 32) {
        warpReduce5(sdata, tid);
    }

    // write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}

template <unsigned int blockSize>
__device__ void warpReduce6(volatile int *sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8)  sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4)  sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2)  sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__global__ void reduce6(int *g_idata, int *g_odata, unsigned int n) {
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + tid;
    unsigned int gridSize = blockSize * 2 * gridDim.x;
    sdata[tid] = 0;

    while (i < n) {
        sdata[tid] += g_idata[i] + g_idata[i + blockSize];
        i += gridSize;
    }
    __syncthreads();

    if (blockSize >= 512) {
        if (tid < 256) {
            sdata[tid] += sdata[tid + 256];
        }
        __syncthreads();
    } 
    if (blockSize >= 256) {
        if (tid < 128) {
            sdata[tid] += sdata[tid + 128];
        }
        __syncthreads();
    } 
    if (blockSize >= 128) {
        if (tid < 64) {
            sdata[tid] += sdata[tid + 64];
        }
        __syncthreads();
    } 

    if (tid < 32) {
        warpReduce6<blockSize>(sdata, tid);
    }

    // write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}

