#include <stdio.h>
#include <hip/hip_runtime.h>

#define cudaCheck(msg)                                                   \
    do {                                                                 \
        hipError_t __err = hipGetLastError();                          \
        if (__err != hipSuccess) {                                      \
            fprintf(stderr, "False error: %s (%s at %s:%d)\n",           \
                    msg, hipGetErrorString(__err), __FILE__, __LINE__); \
            fprintf(stderr, "*** Failed - Aborting\n");                  \
        }                                                                \
    } while (0)

const size_t N = 33ULL * 1024ULL * 1024ULL;
const size_t BLOCK_SIZE = 1024;

__global__ void atomic_red(const float *gdata, float *out) {
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < N) {
        atomicAdd(out, gdata[idx]);
    }
}

int main() {
    float *h_A, *h_sum, *d_A, *d_sum;
    h_A = new float[N];
    h_sum = new float;
    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
    }
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_sum, sizeof(float));

    cudaCheck("hipMalloc failure");

    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    cudaCheck("hipMemcpy H2D failure");

    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_sum, 0, sizeof(float));
    cudaCheck("hipMemset failure");

    atomic_red<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_A, d_sum);
    cudaCheck("atomic reduction kernel launch failure");
    hipMemcpy(h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_sum);
    free(h_A);
    free(h_sum);

    return 0;
}
