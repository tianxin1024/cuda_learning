// link: https://github.com/NVIDIA-developer-blog/code-samples/blob/master/series/cuda-cpp/coalescing-global/coalescing.cu

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess) {
        fprintf(strerr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
        assert(result == cudaSuccess);
    }
#endif
    return result;
}

template <typename T>
__global__ void offset(T* a, int s) {
    int i = blockDim.x * blockIdx.x + threadIdx.x + s;
    a[i] = a[i] + 1;
}

template <typename T>
__global__ void stride(T* a, int s) {
    int i = (blockDim.x * blockIdx.x + threadIdx.x) * s;
    a[i] = a[i] + 1;
}


template <typename T>
void runTest(int deviceId, int nMB) {
    int blockSize = 256;
    float ms;

    T *d_a;
    hipEvent_t startEvent, stopEvent;

    int n = nMB * 1024 * 1024 / sizeof(T);

    checkCuda(hipMalloc(&d_a, n * 33 * sizeof(T)));

    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));

    printf("Offset, Bandwidth (GB/s):\n");

    offset<<<n / blockSize, blockSize>>>(d_a, 0);  // warm up

    for (int i = 1; i <= 32; i++) {
        checkCuda(hipMemset(d_a, 0, n * sizeof(T)));

        checkCuda(hipEventRecord(startEvent, 0));
        offset<<<n / blockSize, blockSize>>>(d_a, i);
        checkCuda(hipEventRecord(stopEvent, 0));
        checkCuda(hipEventSynchronize(stopEvent));

        checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
        // Bandwidth:  (1024 * 1024    *    4    *   2) / (1024 * 1024 * 1024) / ms * 1000
        //                [matrix]       [float]   [r/w]    [1024^3 or 10^9]   /     [1s=1000ms]
        printf("%d, %f\n", i, 2 * nMB / ms);
    }

    printf("\n");
    printf("Stride, Bandwidth (GB/s):\n");

    stride<<<n / blockSize, blockSize>>>(d_a, 1);  // warm up
    for (int i = 1; i <= 32; ++i) {
        checkCuda(hipMemset(d_a, 0, n * sizeof(T)));

        checkCuda(hipEventRecord(startEvent, 0));
        stride<<<n / blockSize, blockSize>>>(d_a, i);
        checkCuda(hipEventRecord(stopEvent, 0));
        checkCuda(hipEventSynchronize(stopEvent));

        checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
        printf("%d, %f\n", i, 2 * nMB / ms);
    }

    checkCuda(hipEventDestroy(startEvent));
    checkCuda(hipEventDestroy(stopEvent));
    hipFree(d_a);
}

int main(int argc, char **argv) {

    int nMB = 4;
    int deviceId = 0;
    bool bFp64 = false;

    for (int i = 1; i < argc; i++) {
        if (!strncmp(argv[i], "dev=", 4)) {
            deviceId = atoi((char*)(&argv[i][4]));
        } else if (!strcmp(argv[i], "fp64")) {
            bFp64 = true;
        }
    }

    hipDeviceProp_t prop;

    checkCuda(hipSetDevice(deviceId));
    checkCuda(hipGetDeviceProperties(&prop, deviceId));
    printf("Device: %s\n", prop.name);
    printf("Transfer size (MB): %d\n", nMB);
    printf("%s Precision\n", bFp64 ? "Double" : "Single");

    if (bFp64) {
        runTest<double>(deviceId, nMB);
    } else {
        runTest<float>(deviceId, nMB);
    }

    return 0;
}
